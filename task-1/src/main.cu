#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <vector>

#include "parameters.hpp"
#include "playground/matmul.hpp"
#include "playground/static.hpp"
#include "playground/utils.hpp"

int main()
{
    auto A = std::vector<params::DataType>(params::M * params::K);
    playground::initRandMat(params::M, params::K, A.data());
    auto B = std::vector<params::DataType>(params::K * params::N);
    playground::initRandMat(params::K, params::N, B.data());
    auto C = std::vector<params::DataType>(params::M * params::N);

    // Gound Truth of C
    auto C_gt = C;
    printf("[Playground] Start Calculating Ground Truth ... ");
    fflush(stdout);
    playground::matmul<params::DataType, 0>(params::M, params::N, params::K,
                                            A.data(), B.data(), C_gt.data());
    printf("Finished!\n");

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    params::DataType *d_A, *d_B, *d_C;
    hipMalloc((void**) &d_A, A.size() * sizeof(params::DataType));
    hipMalloc((void**) &d_B, B.size() * sizeof(params::DataType));
    hipMalloc((void**) &d_C, C.size() * sizeof(params::DataType));

    hipMemcpy(d_A, A.data(), A.size() * sizeof(params::DataType),
               hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data(), B.size() * sizeof(params::DataType),
               hipMemcpyHostToDevice);

    float runtime = 0.0f, sumRuntime = 0.0f;

    printf(
        "[Playground] Start Testing for GEMM Version %d with DType %s ... \n",
        params::MatmulVersion, params::DataTypeName.data());
    // If not using cblas, execute the function multiple times to get average
    // runtime
    if constexpr (params::MatmulVersion != playground::CBLAS_VER) {
        for (auto i = 0ULL; i < params::NumRep + params::NumWarmup; ++i) {
            // Warm Up
            if (i < params::NumWarmup) {
                playground::matmul<params::DataType,
                                   uint8_t(params::MatmulVersion)>(
                    params::M, params::N, params::K, d_A, d_B, d_C);
                continue;
            }
            if (i == params::NumWarmup) {
                printf("[Playground] Warming Up Finished!\n");
            }

            hipEventRecord(start, nullptr);
            playground::matmul<params::DataType, int8_t(params::MatmulVersion)>(
                params::M, params::N, params::K, d_A, d_B, d_C);
            hipEventRecord(stop, nullptr);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&runtime, start, stop);
            sumRuntime += runtime;
        }
        hipMemcpy(C.data(), d_C, C.size() * sizeof(params::DataType),
                   hipMemcpyDeviceToHost);
    }
    // If using cblas, run the function only once
    else {
        hipEventRecord(start, nullptr);
        playground::matmul<params::DataType, uint8_t(params::MatmulVersion)>(
            params::M, params::N, params::K, A.data(), B.data(), C.data());
        hipEventRecord(stop, nullptr);
        hipEventElapsedTime(&runtime, start, stop);
        sumRuntime += runtime;
    }

    hipDeviceSynchronize();
    printf("[Playground] Calculating Finished\n");

    auto avgErr =
        playground::compareMat(params::M, params::N, C_gt.data(), C.data());

    // calculate tflops and average error
    float msecPerMatrixMul = sumRuntime / params::NumRep;
    double flopsPerMatrixMul = 2.0 * params::M * params::N * params::K;
    double tflops =
        (flopsPerMatrixMul * 1.0e-12f) / (msecPerMatrixMul / 1000.0f);

    printf("[Playground] Result >>> TFLOPS: %lf; Average Error: %f\n", tflops,
           avgErr);

    // free memories in device
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}