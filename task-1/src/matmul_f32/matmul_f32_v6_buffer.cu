#include "hip/hip_runtime.h"
// @file: ./task-1/src/f16-v2.cu
#include "playground/matmul.hpp"
#include <cstring>
#include <hip/hip_runtime.h>

namespace playground
{
#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])
#define FLOAT4_CONST(pointer) (reinterpret_cast<const float4*>(&(pointer))[0])
__global__ void matmul_v6(const float* A, const float* B, float* C, int M,
                          int N, int K)
{       const int BM = 128;
        const int BN = 128;
        const int BK = 8;
        const int TM = 8;
        const int TN = 8;

        const int bx = blockIdx.x;
        const int by = blockIdx.y;
        const int tx = threadIdx.x;
        const int ty = threadIdx.y;
        const int tid = ty * blockDim.x + tx;

        __shared__ float s_a[2][BK][BM];
        __shared__ float s_b[2][BK][BN];

        float r_load_a[4];
        float r_load_b[4];
        float r_comp_a[TM];
        float r_comp_b[TN];
        float r_c[TM][TN] = {0.0};

        int load_a_smem_m = tid >> 1;
        int load_a_smem_k = (tid & 1) << 2;
        int load_b_smem_k = tid >> 5;
        int load_b_smem_n = (tid & 31) << 2;

        int load_a_gmem_m = by * BM + load_a_smem_m;
        int load_b_gmem_n = bx * BN + load_b_smem_n;

        {
            int load_a_gmem_k = load_a_smem_k;
            int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_gmem_k, K);
            int load_b_gmem_k = load_b_smem_k;
            int load_b_gmem_addr = OFFSET(load_b_gmem_k, load_b_gmem_n, N);
            FLOAT4(r_load_a[0]) = FLOAT4_CONST(A[load_a_gmem_addr]);
            FLOAT4(r_load_b[0]) = FLOAT4_CONST(B[load_b_gmem_addr]);

            s_a[0][load_a_smem_k][load_a_smem_m] = r_load_a[0];
            s_a[0][load_a_smem_k + 1][load_a_smem_m] = r_load_a[1];
            s_a[0][load_a_smem_k + 2][load_a_smem_m] = r_load_a[2];
            s_a[0][load_a_smem_k + 3][load_a_smem_m] = r_load_a[3];
            FLOAT4(s_b[0][load_b_smem_k][load_b_smem_n]) = FLOAT4(r_load_b[0]);
        }

        for (int bk = 1; bk < (K + BK - 1) / BK; bk++) {

            int smem_sel = (bk - 1) & 1;
            int smem_sel_next = bk & 1;

            int load_a_gmem_k = bk * BK + load_a_smem_k;
            int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_gmem_k, K);
            int load_b_gmem_k = bk * BK + load_b_smem_k;
            int load_b_gmem_addr = OFFSET(load_b_gmem_k, load_b_gmem_n, N);
            FLOAT4(r_load_a[0]) = FLOAT4_CONST(A[load_a_gmem_addr]);
            FLOAT4(r_load_b[0]) = FLOAT4_CONST(B[load_b_gmem_addr]);

#pragma unroll
            for (int tk = 0; tk < BK; tk++) {
                FLOAT4(r_comp_a[0]) = FLOAT4(s_a[smem_sel][tk][ty * TM / 2]);
                FLOAT4(r_comp_a[4]) =
                    FLOAT4(s_a[smem_sel][tk][ty * TM / 2 + BM / 2]);
                FLOAT4(r_comp_b[0]) = FLOAT4(s_b[smem_sel][tk][tx * TN / 2]);
                FLOAT4(r_comp_b[4]) =
                    FLOAT4(s_b[smem_sel][tk][tx * TN / 2 + BN / 2]);

#pragma unroll
                for (int tm = 0; tm < TM; tm++) {
#pragma unroll
                    for (int tn = 0; tn < TN; tn++) {
                        r_c[tm][tn] += r_comp_a[tm] * r_comp_b[tn];
                    }
                }
            }

            s_a[smem_sel_next][load_a_smem_k][load_a_smem_m] = r_load_a[0];
            s_a[smem_sel_next][load_a_smem_k + 1][load_a_smem_m] = r_load_a[1];
            s_a[smem_sel_next][load_a_smem_k + 2][load_a_smem_m] = r_load_a[2];
            s_a[smem_sel_next][load_a_smem_k + 3][load_a_smem_m] = r_load_a[3];
            FLOAT4(s_b[smem_sel_next][load_b_smem_k][load_b_smem_n]) =
                FLOAT4(r_load_b[0]);

            __syncthreads();
        }

#pragma unroll
        for (int tk = 0; tk < BK; tk++) {
            FLOAT4(r_comp_a[0]) = FLOAT4(s_a[1][tk][ty * TM / 2]);
            FLOAT4(r_comp_a[4]) = FLOAT4(s_a[1][tk][ty * TM / 2 + BM / 2]);
            FLOAT4(r_comp_b[0]) = FLOAT4(s_b[1][tk][tx * TN / 2]);
            FLOAT4(r_comp_b[4]) = FLOAT4(s_b[1][tk][tx * TN / 2 + BN / 2]);

#pragma unroll
            for (int tm = 0; tm < TM; tm++) {
#pragma unroll
                for (int tn = 0; tn < TN; tn++) {
                    r_c[tm][tn] += r_comp_a[tm] * r_comp_b[tn];
                }
            }
        }

#pragma unroll
        for (int i = 0; i < TM / 2; i++) {
            int store_c_gmem_m = by * BM + ty * TM / 2 + i;
            int store_c_gmem_n = bx * BN + tx * TN / 2;
            int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
            FLOAT4(C[store_c_gmem_addr]) = FLOAT4(r_c[i][0]);
            FLOAT4(C[store_c_gmem_addr + BN / 2]) = FLOAT4(r_c[i][4]);
        }
#pragma unroll
        for (int i = 0; i < TM / 2; i++) {
            int store_c_gmem_m = by * BM + BM / 2 + ty * TM / 2 + i;
            int store_c_gmem_n = bx * BN + tx * TN / 2;
            int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
            FLOAT4(C[store_c_gmem_addr]) = FLOAT4(r_c[i + TM / 2][0]);
            FLOAT4(C[store_c_gmem_addr + BN / 2]) = FLOAT4(r_c[i + TM / 2][4]);
        }
    }

PG_MATMUL_SIG(float32_t, 6, M, N, K, A, B, C)
{
    dim3 blocks((N + 127) / 128, (M + 127) / 128, 1);
    dim3 threads(16, 16, 1);
    playground::matmul_v6<<<blocks, threads>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}
}  // namespace playground