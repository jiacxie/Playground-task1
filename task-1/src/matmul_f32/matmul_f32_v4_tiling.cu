#include "hip/hip_runtime.h"
// @file: ./task-1/src/f16-v2.cu
#include <cstring>
#include <hip/hip_runtime.h>
#include "playground/matmul.hpp"

namespace playground
{

template <typename T1, typename T2>
    requires std::is_const_v<T1> || (!std::is_const_v<T2>)
__host__ __device__ T1& rCast(T2& x)
{
    // 将T2类型的引用x的地址转换为T1类型的指针，然后解引用得到T1类型的引用
    return *(reinterpret_cast<T1*>(&x));
}

#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])
__global__ void matmul_v4(const float *A, const float *B, float *C, int M, int N, int K) {
    const int BM = 128;
    const int BN = 128;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;

    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tid = ty * blockDim.x + tx;

    __shared__ __align__(32 * 1024) float s_a[BM][BK];
    __shared__ __align__(32 * 1024) float s_b[BK][BN];

    float r_c[TM][TN] = {0.0};

    int load_a_smem_m = tid >> 1;
    int load_a_smem_k = (tid & 1) << 2;
    int load_b_smem_k = tid >> 5;
    int load_b_smem_n = (tid & 31) << 2;

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    for (int bk = 0; bk < (K + BK - 1) / BK; bk++) {
        int load_a_gmem_k = bk * BK + load_a_smem_k;
        int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_gmem_k, K);
        FLOAT4(s_a[load_a_smem_m][load_a_smem_k]) = rCast<const float4> (A[load_a_gmem_addr]);
        int load_b_gmem_k = bk * BK + load_b_smem_k;
        int load_b_gmem_addr = OFFSET(load_b_gmem_k, load_b_gmem_n, N);
        FLOAT4(s_b[load_b_smem_k][load_b_smem_n]) = rCast<const float4> (B[load_b_gmem_addr]);

        __syncthreads();

#pragma unroll
        for (int k = 0; k < BK; k++) {
#pragma unroll
            for (int m = 0; m < TM; m++) {
#pragma unroll
                for (int n = 0; n < TN; n++) {
                    int comp_a_smem_m = ty * TM + m;
                    int comp_b_smem_n = tx * TN + n;
                    r_c[m][n] += s_a[comp_a_smem_m][k] * s_b[k][comp_b_smem_n];
                }
            }
        }

        __syncthreads();
    }

#pragma unroll
    for (int i = 0; i < TM; i++) {
        int store_c_gmem_m = by * BM + ty * TM + i;
#pragma unroll
        for (int j = 0; j < TN; j += 4) {
            int store_c_gmem_n = bx * BN + tx * TN + j;
            int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
            FLOAT4(C[store_c_gmem_addr]) = FLOAT4(r_c[i][j]);
        }
    }
}

PG_MATMUL_SIG(float32_t, 4, M, N, K, A, B, C)
{
    dim3 blocks((N + 127) / 128, (M + 127) / 128, 1);
    dim3 threads(16, 16, 1);
    playground::matmul_v4<<<blocks, threads>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}
}